#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>

#include "../../common/para.h"
#include "../../common/para.cuh"
#include "kernel.cuh"
#include "runtime.cuh"

__global__ void masterKernel(volatile int *done, volatile int *totalScheTasks, volatile gTaskStruct *gTaskPool)
{
	int warpIdxx = (threadIdx.x / warpSize);
	__shared__ volatile int barID;						 // the ID for bar.sync
	__shared__ volatile int smStartIndx;				 // the start pointer for free memory region of shared memory
	__shared__ volatile int doneCtr[BP_NUM];			 // number of warp in a task
	__shared__ volatile gWarpStruct warpPoolDev[BP_NUM]; // warpPool
	int taskPointer;									 //local pointer of task table
	int taskStartP;										 //global pointer of task table
	__shared__ volatile int barIDArray[syncNum];
	__shared__ volatile int sharedTree[SH_TREE_SIZE]; //shared mem data structure
	__shared__ volatile int warpCtr;
	__shared__ volatile int warpId;
	__shared__ volatile int exit;
	extern __shared__ volatile int shared_mem[];
	int i;
	int threadDone;

	// Init warp pool
	if ((threadIdx.x & 0x1f) != 0)
		warpPoolDev[(threadIdx.x & 0x1f)].exec = 0;
	else
		warpPoolDev[(threadIdx.x & 0x1f)].exec = -1;

	taskPointer = 0;
	exit = 0;
	doneCtr[(threadIdx.x & 0x1f)] = 0;

	__threadfence_block();

	// MTB 中第一个 warp, 也就是 scheduler warp
	// scheduler warp 中的每一个线程负责 tasktable column 中的一个 entry
	if (threadIdx.x < warpSize)
	{
		while (!(*done))
		{
			// 其实就是二维数组转成一维数组的后的下标
			// 行号为 taskPointer，列号为 blockIdx.x
			taskStartP = (taskPointer * BK_NUM + blockIdx.x);
			__threadfence_block();
			// 当前Task不是第一个Task，即readyId不是-1
			if (gTaskPool[taskStartP].readyId != -1 && doneCtr[taskPointer] == 0)
			{
				// 当前Task的上一个Task
				if (gTaskPool[gTaskPool[taskStartP].readyId].done == 1)
				{
					barID = -1;
					smStartIndx = -1;
					doneCtr[taskPointer] = gTaskPool[gTaskPool[taskStartP].readyId].thread *
										   gTaskPool[gTaskPool[taskStartP].readyId].block / warpSize;
					warpCtr = doneCtr[taskPointer];
					warpId = 0;
					//parallel scheduling
					while (1)
					{
						threadDone = 1;
						if (threadIdx.x > 0)
						{
							threadDone = 0;
							if (warpPoolDev[threadIdx.x].exec == 0)
							{
								// warpCtr中保存着还需要调度的warp数量
								if (atomicSub((int *)&warpCtr, 1) > 0)
								{
									warpPoolDev[threadIdx.x].warpId = atomicAdd((int *)&warpId, 1) * warpSize;
									// global pointer of task table
									warpPoolDev[threadIdx.x].bufferNum = gTaskPool[taskStartP].readyId;
									warpPoolDev[threadIdx.x].SMindex = smStartIndx; // shared mem. index
									warpPoolDev[threadIdx.x].barId = barID;			// index of threadblock
																					// num. of thread
									warpPoolDev[threadIdx.x].threadNum = gTaskPool[gTaskPool[taskStartP].readyId].thread;
									warpPoolDev[threadIdx.x].taskId = taskPointer; // local pointer of task table
									__threadfence_block();						   // To make sure the exec. is worked after fence
									warpPoolDev[threadIdx.x].exec = 1;
									__threadfence_block();
								} // End atomic
							}
						}
						// 当前Task所需的warp已经全部分配完了
						if (warpCtr <= 0)
							threadDone = 1;
						// 保证 scheduler warp 中的32个线程都全部执行完毕
						if (__all(threadDone == 1) != 0)
						{
							break;
						}

					} // End while(1)

					//gTaskPool[taskStartP-1].done = 0; // reset flag whenever task scheduling has been done
					gTaskPool[gTaskPool[taskStartP].readyId].done = 0;
					gTaskPool[taskStartP].readyId = -1;

				} // End if ready flag
			}

			// taskPoint即行号，行号++，循环该Column的每一行
			taskPointer++; // renew the local pointer of task table
			if (taskPointer == BP_NUM)
				taskPointer = 0;

		} // End while done
		exit = 1;
		__threadfence_block();
	} // End if thread < 32

#if 1
	// executor warp
	else
	{
		//while(!(*done)){
		while (!exit)
		{
			// 当前 executor warp 的 exec 标志位被设置
			// 意味着该 executor warp 已经被 scheduler warp 安排了任务，需要执行
			if (warpPoolDev[warpIdxx].exec == 1)
			{
				// kernel running here
#if 1
				mult_gpu((float *)gTaskPool[warpPoolDev[warpIdxx].bufferNum].para[0],
						 (float *)gTaskPool[warpPoolDev[warpIdxx].bufferNum].para[1],
						 (float *)gTaskPool[warpPoolDev[warpIdxx].bufferNum].para[2],
						 (int)gTaskPool[warpPoolDev[warpIdxx].bufferNum].para[3],
						 (int)gTaskPool[warpPoolDev[warpIdxx].bufferNum].para[4],
						 warpPoolDev[warpIdxx].warpId);
#endif
				// 每个 executor warp 中的第一个线程负责在当前 warp 执行完毕后，将计数器的值减一
				// 该计数器统计当前 task 未完成的 warp 数量，当计数器的值为0时，该 task 执行完毕
				if ((threadIdx.x & 0x1f) == 0)
				{
					// 如果当前 warp 是该 task 的最后一个warp，那么该 warp 执行完成后要处理一些最后的工作
					if (atomicSub((int *)&doneCtr[warpPoolDev[warpIdxx].taskId], 1) == 1)
					{ // when all warps in a task have been done
						__threadfence_system();
						gTaskPool[warpPoolDev[warpIdxx].bufferNum].ready = 0; //unset the ready flag

						atomicAdd((int *)totalScheTasks, 1); //update the global task counter
					}

					warpPoolDev[warpIdxx].exec = 0;
					__threadfence_block();

				} // End if threadIdx.x
			}	  // End if exec
		}		  // End while done
	}			  // End else
#endif
}
